#include "hip/hip_runtime.h"
/**
 * @file v3.cpp
 * @author GKC_NKCS (2012522@mail.nankai.edu.cn)
 * @brief
 * @version 0.3
 * @date 2022-07-09
 *
 * @copyright Copyright (c) 2022
 * @details mainbody of MPI gauss elimination with streams
 *
 */
#include <stdio.h>
#include <string>
#include <iostream>
#include "file.h"
#include "bitmap.h"
#include "hip/hip_runtime.h"
#include ""
#include "CheckCuda.cuh"

#define streamNumber 5

double s_time;    // start time
double e_time;    // end time
int* eliminatant; // eliminatant wnd
int* eliminator;  // eliminatant wnd
int* tmp;
int* table;
int* row;
int wndSize;      // max cols
int wndSize1;     // rows of eliminatant wnd
int wndSize2;     // rows of eliminator wnd
int wrdLen;       // cols per row
BitManager* eliminatantManager;
BitManager* eliminatorManager;

int threadsPerBlock;
int blocksPerGrid;

string basePath = "F:/大二下课程/并行计算/期末研究报告相关材料/data/Groebner/";
// string basePath = "/home/bill/Desktop/para/src/Groebner/";
string examplePath = basePath + getExampleName(5);

void init();
void gaussian();
void write();

int main(int argc, char* argv[])
{
    float cudaElaspedTime = 0.0;
    hipEvent_t cudaStart;
    hipEvent_t cudaEnd;
    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaEnd);

    getParam(examplePath, wndSize1, wndSize2, wndSize); // get size of wnd

    /* init wnd and relavant params */
    init();

    /* conduct elimination and timing*/
    hipEventRecord(cudaStart, 0);
    gaussian();
    checkCuda(hipEventRecord(cudaEnd, 0));
    checkCuda(hipEventSynchronize(cudaEnd));
    checkCuda(hipEventElapsedTime(&cudaElaspedTime, cudaStart, cudaEnd));
    printf("cuda elasped time: %f ms.\n", cudaElaspedTime);

    /*  gather and output result */
    write();

    for (int i = 0; i < wndSize1; i++)
    {
        delete[] eliminatantManager[i].idx;
        eliminatantManager[i].idx = nullptr;
    }
    for (int i = 0; i < wndSize; i++)
    {
        if (eliminatorManager[i].idx != nullptr)
        {
            delete[] eliminatorManager[i].idx;
            eliminatorManager[i].idx = nullptr;
        }
    }
    delete[] eliminatantManager;
    delete[] eliminatorManager;
    eliminatantManager = nullptr;
    eliminatorManager = nullptr;
    checkCuda(hipFree(eliminatant));
    checkCuda(hipFree(eliminator));
    /*checkCuda(hipFree(tmp));
    checkCuda(hipFree(table));
    checkCuda(hipFree(row));*/
    return 0;
}

__global__ void initWith(int* wnd, int len, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int grid = blockDim.x * gridDim.x;
    int i = tid;
    while (i < len)
    {
        wnd[i] = num;
        i += grid;
    }
}

void initWithCPU(int* wnd, int len, int num)
{
    for (int i = 0; i < len; i++)
    {
        wnd[i] = num;
    }
}

void init()
{
    wrdLen = wndSize / WORD_BITS + 1;

    size_t eliminatantSize = wndSize1 * wrdLen * sizeof(int);
    size_t eliminatorSize = wndSize * wrdLen * sizeof(int);
    checkCuda(hipMallocManaged(&eliminatant, eliminatantSize));
    checkCuda(hipMallocManaged(&eliminator, eliminatorSize));

    int deviceId;
    int numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    threadsPerBlock = 1024;
    blocksPerGrid = 32 * numberOfSMs;

    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminatant, wndSize1 * wrdLen, 0);
    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminator, wndSize * wrdLen, 0);
    checkCuda(hipDeviceSynchronize());

    string* eliminatantSparseWnd = new string[wndSize1];
    getSparseMatrix(examplePath, eliminatantSparseWnd, wndSize1, ELIMINATANT);
    createWnd(eliminatantSparseWnd, eliminatant, wndSize1, wrdLen, UNORDERED);
    delete[] eliminatantSparseWnd;
    eliminatantSparseWnd = nullptr;

    string* eliminatorSparseWnd = new string[wndSize2];
    getSparseMatrix(examplePath, eliminatorSparseWnd, wndSize2, ELIMINATOR);
    createWnd(eliminatorSparseWnd, eliminator, wndSize2, wrdLen, ORDERED);
    delete[] eliminatorSparseWnd;
    eliminatorSparseWnd = nullptr;

    eliminatantManager = new BitManager[wndSize1];
    eliminatorManager = new BitManager[wndSize];
    buildBitManager(eliminatant, wrdLen, eliminatantManager, wndSize1);
    buildBitManager(eliminator, wrdLen, eliminatorManager, wndSize);

    /*size_t tmpSize = wrdLen * sizeof(int);
    size_t tableSize = streamNumber * sizeof(int);
    checkCuda(hipMallocManaged(&tmp, tmpSize));
    checkCuda(hipMallocManaged(&table, tableSize));
    checkCuda(hipMallocManaged(&row, tableSize));
    initWith << <blocksPerGrid, threadsPerBlock >> > (tmp, wrdLen, 0);
    initWith << <blocksPerGrid, threadsPerBlock >> > (table, streamNumber, -1);
    initWith << <blocksPerGrid, threadsPerBlock >> > (row, streamNumber, -1);
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipGetLastError());*/
    tmp = new int[wrdLen];
    table = new int[streamNumber];
    row = new int[streamNumber];
    initWithCPU(tmp, wrdLen, 0);
    initWithCPU(table, streamNumber, -1);
    initWithCPU(row, streamNumber, -1);
}

__global__ void eliminationKernelFunction(int* bitmap1, int* bitmap2, int wrdLen)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int grid = blockDim.x * gridDim.x;
    int i = tid;
    while (i < wrdLen)
    {
        bitmap1[i] ^= bitmap2[i];
        i += grid;
    }
}

void gaussian()
{
    BitManager tmpManager;
    for (int i = 0; i < wndSize1; i++)
    {
        // dealing with result of last row
        buildBitManager(tmp, wrdLen, &tmpManager);
        int tmpLftCol = tmpManager.lftCol;
        if (tmpLftCol != -1 && eliminatorManager[tmpLftCol].lftCol == -1)
        {
            copyBitMap(tmp, eliminator + tmpLftCol * wrdLen, &tmpManager, eliminatorManager + tmpLftCol);
        }
        // dealing with current row
        int lftCol = eliminatantManager[i].lftCol;
        table[0] = lftCol;
        row[0] = i;
        while (lftCol != -1 && eliminatorManager[lftCol].lftCol != -1)
        {
            int cnt = 1;
            for (int j = i + 1; j < wndSize1 && cnt < streamNumber; j++)
            {
                int tmpLftCol = eliminatantManager[j].lftCol;
                if (tmpLftCol != -1 && eliminatantManager[tmpLftCol].lftCol != -1)
                {
                    table[cnt] = tmpLftCol;
                    row[cnt] = j;
                    //cout << "cnt: " << cnt << " j: "<<j<<" table[cnt]: " << table[cnt] << " row[cnt]: " << row[cnt]<<endl;
                    cnt++;
                }
            }
            for (int j = 0; j < cnt; ++j)
            {
                hipStream_t stream;
                checkCuda(hipStreamCreate(&stream));
                //cout << "j = " << j << " row[j] " << row[j] << " table[j] " << table[j] << endl;
                eliminationKernelFunction << <blocksPerGrid, threadsPerBlock >> > (eliminatant + row[j] * wrdLen, eliminator + table[j] * wrdLen, wrdLen);
                checkCuda(hipStreamDestroy(stream));
            }
            checkCuda(hipDeviceSynchronize());
            checkCuda(hipGetLastError());
            for (int j = 0; j < cnt; j++)
            {
                buildBitManager(eliminatant + row[j] * wrdLen, wrdLen, eliminatantManager + row[j]);
            }
            lftCol = eliminatantManager[i].lftCol;
            table[0] = lftCol;
        }

        // store the result of current row
        copyBitmapSingle(eliminatant + i * wrdLen, tmp, wrdLen);
    }
}

void write()
{
    string* result = new string[wndSize1];
    toString(eliminatant, wrdLen, result, wndSize1);
    writeResult(examplePath, result, wndSize1);
    delete[] result;
    result = nullptr;
}