#include "hip/hip_runtime.h"
/**
 * @file v2.cpp
 * @author GKC_NKCS (2012522@mail.nankai.edu.cn)
 * @brief
 * @version 0.2
 * @date 2022-07-09
 *
 * @copyright Copyright (c) 2022
 * @details mainbody of CUDA gauss elimination
 *
 */
#include <stdio.h>
#include <string>
#include <iostream>
#include "file.h"
#include "bitmap.h"
#include "hip/hip_runtime.h"
#include ""
#include "CheckCuda.cuh"

double s_time;    // start time
double e_time;    // end time
int* eliminatant; // eliminatant wnd
int* eliminator;  // eliminatant wnd
int wndSize;      // max cols
int wndSize1;     // rows of eliminatant wnd
int wndSize2;     // rows of eliminator wnd
int wrdLen;       // cols per row
BitManager* eliminatantManager;
BitManager* eliminatorManager;
int* table1;
int* table2;

int threadsPerBlock;
int blocksPerGrid;

string basePath = "F:/大二下课程/并行计算/期末研究报告相关材料/data/Groebner/";
//string basePath = "/home/bill/Desktop/para/src/Groebner/";
string examplePath = basePath + getExampleName(1);

void init();
void gaussian();
void write();

int main(int argc, char* argv[])
{
    float cudaElaspedTime = 0.0;
    hipEvent_t cudaStart;
    hipEvent_t cudaEnd;
    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaEnd);

    getParam(examplePath, wndSize1, wndSize2, wndSize); // get size of wnd

    /* init wnd and relavant params */
    init();

    /* conduct elimination and timing*/
    hipEventRecord(cudaStart, 0);
    gaussian();
    checkCuda(hipEventRecord(cudaEnd, 0));
    checkCuda(hipEventSynchronize(cudaEnd));
    checkCuda(hipEventElapsedTime(&cudaElaspedTime, cudaStart, cudaEnd));
    printf("cuda elasped time: %f ms.\n", cudaElaspedTime);

    /*  gather and output result */
    write();

    delete[] eliminatantManager;
    delete[] eliminatorManager;
    eliminatantManager = nullptr;
    eliminatorManager = nullptr;
    checkCuda(hipFree(eliminatant));
    checkCuda(hipFree(eliminator));
    checkCuda(hipFree(table1));
    checkCuda(hipFree(table2));
    return 0;
}

__global__ void initWith(int* wnd, int len, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int grid = blockDim.x * gridDim.x;
    int i = tid;
    while (i<len)
    {
        wnd[i] = num;
        i += grid;
    }
}

void init()
{
    wrdLen = wndSize / WORD_BITS + 1;

    size_t eliminatantSize = wndSize1 * wrdLen * sizeof(int);
    size_t eliminatorSize = wndSize * wrdLen * sizeof(int);
    checkCuda(hipMallocManaged(&eliminatant, eliminatantSize));
    checkCuda(hipMallocManaged(&eliminator, eliminatorSize));

    int deviceId;
    int numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    threadsPerBlock = 1024;
    blocksPerGrid = 32 * numberOfSMs;

    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminatant, wndSize1 * wrdLen, 0);
    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminator, wndSize * wrdLen, 0);
    checkCuda(hipDeviceSynchronize());

    string* eliminatantSparseWnd = new string[wndSize1];
    getSparseMatrix(examplePath, eliminatantSparseWnd, wndSize1, ELIMINATANT);
    createWnd(eliminatantSparseWnd, eliminatant, wndSize1, wrdLen, UNORDERED);
    delete[] eliminatantSparseWnd;
    eliminatantSparseWnd = nullptr;

    string* eliminatorSparseWnd = new string[wndSize2];
    getSparseMatrix(examplePath, eliminatorSparseWnd, wndSize2, ELIMINATOR);
    createWnd(eliminatorSparseWnd, eliminator, wndSize2, wrdLen, ORDERED);
    delete[] eliminatorSparseWnd;
    eliminatorSparseWnd = nullptr;

    eliminatantManager = new BitManager[wndSize1];
    eliminatorManager = new BitManager[wndSize];
    buildBitManager(eliminatant, wrdLen, eliminatantManager, wndSize1);
    buildBitManager(eliminator, wrdLen, eliminatorManager, wndSize);

    size_t table1Size = wndSize1 * sizeof(int);
    size_t table2Size = wndSize * sizeof(int);
    checkCuda(hipMallocManaged(&table1, table1Size));
    checkCuda(hipMallocManaged(&table2, table2Size));
    initWith<<<blocksPerGrid,threadsPerBlock>>>(table1, wndSize1, -1);
    initWith<<<blocksPerGrid,threadsPerBlock>>>(table2, wndSize, -1);
    checkCuda(hipDeviceSynchronize());
}

__global__ 
void eliminationKernelFunction(int* eliminatant, int* eliminator, int* table1, int* table2, int rank, int wndSize1, int wrdLen)
{
    __shared__ int correspondingRow;
    int row = blockIdx.x + rank;
    int i = row;
    int j;
    while (i < wndSize1)
    {
        if(threadIdx.x == 0)
            correspondingRow = table1[i];
        __syncthreads();
        if (correspondingRow != -1)
        {
            for(j = threadIdx.x;j<wrdLen;j+=blockDim.x)
            {
                eliminatant[i * wrdLen + j] ^= eliminator[correspondingRow * wrdLen + j];
            }
        }
        if (threadIdx.x == 0)
        {
            table1[i] = -1;
            int flag = 0b10000000000000000000000000000000;
            bool out = true;
            for (int wrdIdx = wrdLen - 1; out && wrdIdx >= 0; wrdIdx--) // scan from tail to head
            {
                int tmp = *(eliminatant + i * wrdLen + wrdIdx);
                if (tmp != 0)
                {
                    for (int bitIdx = WORD_BITS - 1; bitIdx >= 0; bitIdx--, tmp <<= 1)
                    {
                        if ((tmp & flag) == 0)  continue;
                        table1[i] = wrdIdx * WORD_BITS + bitIdx;
                        out = false;
                        break;
                    }
                }
            }
            correspondingRow = table1[i];
        }
        __syncthreads();
        if (correspondingRow == -1 || table2[correspondingRow] == -1)
        {
            i += gridDim.x;
        }
    }
}

void gaussian()
{
    int i, j;
    int* tmp = new int[wrdLen] {0};
    BitManager tmpManager;
    for (i = 0; i < wndSize1; i++)
    {
        buildBitManager(tmp, wrdLen, &tmpManager);

        int tmpLftCol = tmpManager.lftCol;
        if (tmpLftCol != -1 && eliminatorManager[tmpLftCol].lftCol == -1)
        {
            copyBitMap(tmp, eliminator + tmpLftCol * wrdLen, &tmpManager, eliminatorManager + tmpLftCol);
        }

        for (int idx = 0; idx < wndSize; idx++)
        {
            table2[idx] = eliminatorManager[idx].lftCol;
        }

        for (j = i; j < wndSize1; j++)
        {
            int lftCol = eliminatantManager[j].lftCol;
            table1[j] = eliminatorManager[lftCol].lftCol == -1 ? -1 : lftCol;
        }
        eliminationKernelFunction<<<blocksPerGrid, threadsPerBlock, 64>>>(eliminatant, eliminator, table1, table2, i, wndSize1, wrdLen);
        checkCuda(hipDeviceSynchronize());
        checkCuda(hipGetLastError());

        buildBitManager(eliminatant + wrdLen * i, wrdLen, eliminatantManager + i, wndSize1 - i);
        copyBitmapSingle(eliminatant+i*wrdLen, tmp, wrdLen);
    }
}

void write()
{
    string* result = new string[wndSize1];
    toString(eliminatant, wrdLen, result, wndSize1);
    writeResult(examplePath, result, wndSize1);
}