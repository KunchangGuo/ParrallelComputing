#include "hip/hip_runtime.h"
/**
 * @file v1.cpp
 * @author GKC_NKCS (2012522@mail.nankai.edu.cn)
 * @brief
 * @version 0.1
 * @date 2022-07-06
 *
 * @copyright Copyright (c) 2022
 * @details mainbody of MPI gauss elimination
 *
 */
#include <stdio.h>
#include <string>
#include <iostream>
#include "file.h"
#include "bitmap.h"
#include "hip/hip_runtime.h"
#include ""
#include "CheckCuda.cuh"

double s_time;    // start time
double e_time;    // end time
int* eliminatant; // eliminatant wnd
int* eliminator;  // eliminatant wnd
int wndSize;      // max cols
int wndSize1;     // rows of eliminatant wnd
int wndSize2;     // rows of eliminator wnd
int wrdLen;       // cols per row
BitManager* eliminatantManager;
BitManager* eliminatorManager;

int threadsPerBlock;
int blocksPerGrid;

string basePath = "F:/大二下课程/并行计算/期末研究报告相关材料/data/Groebner/";
//string basePath = "/home/bill/Desktop/para/src/Groebner/";
string examplePath = basePath + getExampleName(4);

void init();
void gaussian();
void write();

int main(int argc, char* argv[])
{
    float cudaElaspedTime = 0.0;
    hipEvent_t cudaStart;
    hipEvent_t cudaEnd;
    hipEventCreate(&cudaStart);
    hipEventCreate(&cudaEnd);

    getParam(examplePath, wndSize1, wndSize2, wndSize); // get size of wnd

    /* init wnd and relavant params */
    init();

    /* conduct elimination and timing*/
    hipEventRecord(cudaStart, 0);
    gaussian();
    checkCuda(hipEventRecord(cudaEnd, 0));
    checkCuda(hipEventSynchronize(cudaEnd));
    checkCuda(hipEventElapsedTime(&cudaElaspedTime, cudaStart, cudaEnd));
    printf("cuda elasped time: %f ms.\n", cudaElaspedTime);

    /*  gather and output result */
    write();

    delete[] eliminatantManager;
    delete[] eliminatorManager;
    eliminatantManager = nullptr;
    eliminatorManager = nullptr;
    checkCuda(hipFree(eliminatant));
    checkCuda(hipFree(eliminator));
    return 0;
}

__global__ void initWith(int* wnd, int len, int num)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int grid = blockDim.x * gridDim.x;
    int i = tid;
    while (i<len)
    {
        wnd[i] = num;
        i += grid;
    }
}

void init()
{
    wrdLen = wndSize / WORD_BITS + 1;

    size_t eliminatantSize = wndSize1 * wrdLen * sizeof(int);
    size_t eliminatorSize = wndSize * wrdLen * sizeof(int);
    checkCuda(hipMallocManaged(&eliminatant, eliminatantSize));
    checkCuda(hipMallocManaged(&eliminator, eliminatorSize));

    int deviceId;
    int numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    threadsPerBlock = 1024;
    blocksPerGrid = 32 * numberOfSMs;

    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminatant, wndSize1 * wrdLen, 0);
    initWith << <blocksPerGrid, threadsPerBlock >> > (eliminator, wndSize * wrdLen, 0);
    checkCuda(hipDeviceSynchronize());

    string* eliminatantSparseWnd = new string[wndSize1];
    getSparseMatrix(examplePath, eliminatantSparseWnd, wndSize1, ELIMINATANT);
    createWnd(eliminatantSparseWnd, eliminatant, wndSize1, wrdLen, UNORDERED);
    delete[] eliminatantSparseWnd;
    eliminatantSparseWnd = nullptr;

    string* eliminatorSparseWnd = new string[wndSize2];
    getSparseMatrix(examplePath, eliminatorSparseWnd, wndSize2, ELIMINATOR);
    createWnd(eliminatorSparseWnd, eliminator, wndSize2, wrdLen, ORDERED);
    delete[] eliminatorSparseWnd;
    eliminatorSparseWnd = nullptr;

    eliminatantManager = new BitManager[wndSize1];
    eliminatorManager = new BitManager[wndSize];
    buildBitManager(eliminatant, wrdLen, eliminatantManager, wndSize1);
    buildBitManager(eliminator, wrdLen, eliminatorManager, wndSize);
}

__global__ 
void eliminationKernelFunction(int* bitmap1, int* bitmap2, int wrdLen)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int grid = blockDim.x * gridDim.x;
    int i = tid;
    while (i < wrdLen)
    {
        bitmap1[i] ^= bitmap2[i];
        i += grid;
    }
}

void gaussian()
{
    int* tmp = new int[wrdLen] {0};
    BitManager tmpManager;
    for (int i = 0; i < wndSize1; i++)
    {
        // dealing with result of last row
        buildBitManager(tmp, wrdLen, &tmpManager);
        int tmpLftCol = tmpManager.lftCol;
        if (tmpLftCol != -1 && eliminatorManager[tmpLftCol].lftCol == -1)
        {
            copyBitMap(tmp, eliminator + tmpLftCol * wrdLen, &tmpManager, eliminatorManager + tmpLftCol);
        }
        // dealing with current row
        int lftCol = eliminatantManager[i].lftCol;
        while (lftCol != -1 && eliminatorManager[lftCol].lftCol != -1)
        {
            eliminationKernelFunction << <blocksPerGrid, threadsPerBlock >> > (eliminatant + i * wrdLen, eliminator + lftCol * wrdLen, wrdLen);
            checkCuda(hipDeviceSynchronize());
            checkCuda(hipGetLastError());
            buildBitManager(eliminatant + i * wrdLen, wrdLen, eliminatantManager + i);
            lftCol = eliminatantManager[i].lftCol;
        }
        // store the result of current row
        copyBitmapSingle(eliminatant+i*wrdLen, tmp, wrdLen);
    }
}

void write()
{
    string* result = new string[wndSize1];
    toString(eliminatant, wrdLen, result, wndSize1);
    writeResult(examplePath, result, wndSize1);
}